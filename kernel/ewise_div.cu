#include "hip/hip_runtime.h"
#include "ewise_div.cuh"

namespace needle {

/* EwiseAdd */
template<typename Dtype>
static __global__ void vec_div(Dtype *a, Dtype *b, Dtype *c, size_t n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] / b[idx];
}

template<typename Dtype>
void vec_div_wrapper(Dtype *a, Dtype *b, Dtype *c, size_t n) {
    const int block_size = 256;
    const int grid_size = (n + block_size - 1) / block_size;
    vec_div<<<grid_size, block_size>>>(a, b, c, n);
}

template __global__ void vec_div<float>(float *a, float *b, 
                                        float *c, size_t n);

template void vec_div_wrapper<float>(float *a, float *b, 
                                     float *c, size_t n);

} //namespace needle

