#include "hip/hip_runtime.h"
#include "ewise_mul.cuh"

namespace needle {

/* EwiseAdd */
template<typename Dtype>
static __global__ void vec_mul(Dtype *a, Dtype *b, Dtype *c, size_t n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] * b[idx];
}

template<typename Dtype>
void vec_mul_wrapper(Dtype *a, Dtype *b, Dtype *c, size_t n) {
    const int block_size = 256;
    const int grid_size = (n + block_size - 1) / block_size;
    vec_mul<<<grid_size, block_size>>>(a, b, c, n);
}

template __global__ void vec_mul<float>(float *a, float *b, 
                                        float *c, size_t n);
template void vec_mul_wrapper<float>(float *a, float *b, 
                                     float *c, size_t n);

} //namespace needle

